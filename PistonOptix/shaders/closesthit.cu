#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"
#include "PistonOptix/inc/LightParameters.h"


// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );
rtDeclareVariable(ShadowPRD, prd_shadow, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, parMaterialIndex, , ); // Per Material index into the sysMaterialParameters array.
rtDeclareVariable(int, programId, , );

rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFEval;

rtBuffer< rtCallableProgramId<void(LightParameter &light, PerRayData &prd, LightSample &sample)> > sysLightSample;
rtBuffer<LightParameter> sysLightParameters;

RT_FUNCTION float sdot(float3 x, float3 y)
{
	return clamp(dot(x, y), 0.0f, 1.0f);
}

RT_FUNCTION float SmoothnessToPhongAlpha(float s)
{
	return pow(1000.0f, s * s);
}


RT_PROGRAM void closesthit()
{
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	// Advance the path to the hit position in world coordinates.
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance; 

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		shading_normal = -shading_normal;
	}

	State state;
	state.hit_position = thePrd.hit_pos;
	state.shading_normal = shading_normal;

	// A material system with support for arbitrary mesh lights would evaluate its emission here.
	thePrd.radiance = make_float3(0.0f);

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	MaterialParameter mat = sysMaterialParameters[parMaterialIndex];

	float3 baseColor = mat.albedo;
	float metallic = mat.metallic;

	float3 diffuseBRDF = make_float3(0.0f);
	float3 specularBRDF = make_float3(0.0f);

	float diffChance = intensity(baseColor);

	// Roulette-select the ray's path
	float roulette = rng(thePrd.seed);
	if (roulette < diffChance) 
	{
		// Diffuse reflection
		sysBRDFSample[EBrdfTypes::LAMBERT](mat, state, thePrd);
		sysBRDFPdf[EBrdfTypes::LAMBERT](mat, state, thePrd);
		diffuseBRDF = sysBRDFEval[EBrdfTypes::LAMBERT](mat, state, thePrd);
	}
	else
	{
		// Specular reflection
		sysBRDFSample[EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		sysBRDFPdf[EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		specularBRDF = sysBRDFEval[EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
	}

	float3 wiWorld = thePrd.wi;
	float3 woWorld = -theRay.direction;
	float3 H = normalize(woWorld + wiWorld);

	float3 dielectricSpecular = make_float3(0.04f, 0.04f, 0.04f);
	float3 F0 = lerp(dielectricSpecular, baseColor, metallic);
	float3 F = F0 + (1.0f - F0) * powf(1.0f - dot(wiWorld, H), 5.0f);

	float3 f = (1.0f - F) * diffuseBRDF + specularBRDF;
	

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	
	thePrd.f_over_pdf = f * fabsf(optix::dot(thePrd.wi, state.shading_normal)) / thePrd.pdf;

	// Add direct light sample weighted by shadow term and 1/probability.
	// The pdf for a directional area light is 1/solid_angle.

	const LightParameter& light = sysLightParameters[0];
	const float3 light_center = state.hit_position + light.direction;
	const float r1 = rng(thePrd.seed);
	const float r2 = rng(thePrd.seed);
	const float2 disk_sample = square_to_disk(make_float2(r1, r2));
	const float3 jittered_pos = light_center + light.radius*disk_sample.x*light.u + light.radius*disk_sample.y*light.v;
	const float3 L = normalize(jittered_pos - state.hit_position);

	const float NdotL = dot(state.shading_normal, L);
	if (NdotL > 0.0f) 
	{
		ShadowPRD shadow_prd;
		shadow_prd.attenuation = make_float3(1.0f);

		optix::Ray shadow_ray(state.hit_position, L, /*shadow ray type*/ 1, 0.0f);
		rtTrace(sysTopObject, shadow_ray, shadow_prd);

		const float solid_angle = light.radius*light.radius*M_PIf;
		thePrd.radiance += NdotL * light.emission * 0.001f * solid_angle * shadow_prd.attenuation;
	}
}

RT_PROGRAM void any_hit()
{
	prd_shadow.attenuation = make_float3(0.0f);
	rtTerminateRay();
	
	thePrd.flags |= FLAG_TERMINATE;
}
