#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, parMaterialIndex, , ); // Per Material index into the sysMaterialParameters array.
rtDeclareVariable(int, programId, , );

rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFEval;


RT_FUNCTION float sdot(float3 x, float3 y)
{
	return clamp(dot(x, y), 0.0f, 1.0f);
}

RT_FUNCTION float SmoothnessToPhongAlpha(float s)
{
	return pow(1000.0f, s * s);
}


RT_PROGRAM void closesthit()
{
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	// Advance the path to the hit position in world coordinates.
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance; 

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		shading_normal = -shading_normal;
		// Do not recalculate the frontface condition!
	}

	State state;
	state.hit_position = thePrd.hit_pos;
	state.shading_normal = shading_normal;

	// A material system with support for arbitrary mesh lights would evaluate its emission here.
	thePrd.radiance = make_float3(0.0f);

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	MaterialParameter mat = sysMaterialParameters[parMaterialIndex];


	float3 f = make_float3(0.0f, 0.0f, 0.0f);
	mat.albedo = fminf(1.0f - mat.specular, mat.albedo);
	float specChance = intensity(mat.specular);
	float diffChance = intensity(mat.albedo);

	// Roulette-select the ray's path
	float roulette = rng(thePrd.seed);
	if (roulette < diffChance)
	{
		// Diffuse reflection
		sysBRDFSample[0](mat, state, thePrd);
		sysBRDFPdf[0](mat, state, thePrd);
		f = sysBRDFEval[0](mat, state, thePrd) / diffChance;
	}
	else if(roulette < specChance + diffChance)
	{
		mat.roughness = SmoothnessToPhongAlpha(1 - mat.roughness);

		// Specular reflection
		sysBRDFSample[1](mat, state, thePrd);
		sysBRDFPdf[1](mat, state, thePrd);
		f = sysBRDFEval[1](mat, state, thePrd) / specChance;
	}

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	
	thePrd.f_over_pdf = f * fabsf(optix::dot(thePrd.wi, state.shading_normal)) / thePrd.pdf;
}
