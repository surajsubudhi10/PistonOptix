#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"
#include "PistonOptix/inc/LightParameters.h"


// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );
rtDeclareVariable(ShadowPRD, prd_shadow, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent, attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord, attribute TEXCOORD, );

// POptix::Material parameter definition.
rtBuffer<POptix::Material> sysMaterialParameters; // Context global buffer with an array of structures of POptix::Material.
rtDeclareVariable(int, parMaterialIndex, , ); // Per POptix::Material index into the sysMaterialParameters array.
rtDeclareVariable(int, programId, , );

rtBuffer< rtCallableProgramId<void(POptix::Material &mat, State &state, PerRayData &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(POptix::Material &mat, State &state, PerRayData &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(POptix::Material &mat, State &state, PerRayData &prd)> > sysBRDFEval;

rtBuffer< rtCallableProgramId<void(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)> > sysLightSample;
rtBuffer<POptix::Light> sysLightParameters;
rtDeclareVariable(int, sysNumberOfLights, , );

RT_FUNCTION float sdot(float3 x, float3 y)
{
	return clamp(dot(x, y), 0.0f, 1.0f);
}

RT_FUNCTION float SmoothnessToPhongAlpha(float s)
{
	return pow(1000.0f, s * s);
}


RT_PROGRAM void closesthit()
{
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	// Advance the path to the hit position in world coordinates.
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance;

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		shading_normal = -shading_normal;
	}

	State state;
	state.hit_position = thePrd.hit_pos;
	state.shading_normal = shading_normal;
	state.geometry_normal = geoNormal;

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.radiance = make_float3(0.0f);
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	POptix::Material mat = sysMaterialParameters[parMaterialIndex];
	float3 baseColor = mat.albedo;
	float metallic = mat.metallic;

	float3 diffuseBRDF = make_float3(0.0f);
	float3 specularBRDF = make_float3(0.0f);

	// Roulette-select the ray's path
	float roulette = rng(thePrd.seed);
	float diffChance = 0.5f * (1.0f - mat.metallic);
	if (roulette < diffChance)
	{
		// Diffuse reflection
		sysBRDFSample[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
		sysBRDFPdf[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
		diffuseBRDF = sysBRDFEval[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);

		thePrd.brdf_flags |= POptix::BSDF_REFLECTION;
		thePrd.brdf_flags |= POptix::BSDF_DIFFUSE;
	}
	else
	{
		// Specular reflection
		sysBRDFSample[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		sysBRDFPdf[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		specularBRDF = sysBRDFEval[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);

		thePrd.brdf_flags |= POptix::BSDF_REFLECTION;
		thePrd.brdf_flags |= (mat.roughness > 0.0f) ? POptix::BSDF_GLOSSY : POptix::BSDF_SPECULAR;
	}

	float3 wiWorld = thePrd.wi;
	float3 woWorld = -theRay.direction;
	float3 H = normalize(woWorld + wiWorld);

	float3 dielectricSpecular = make_float3(0.04f, 0.04f, 0.04f);
	float3 F0 = lerp(dielectricSpecular, baseColor, metallic);
	float3 F = F0 + (1.0f - F0) * powf(1.0f - dot(wiWorld, H), 5.0f);
	float3 f = (1.0f - F) * diffuseBRDF + specularBRDF;

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	thePrd.f_over_pdf = f * fabsf(optix::dot(thePrd.wi, state.shading_normal)) / thePrd.pdf;

#if USE_NEXT_EVENT_ESTIMATION
	if ((thePrd.brdf_flags & (POptix::BSDF_DIFFUSE | POptix::BSDF_GLOSSY)) && 0 < sysNumberOfLights)
	{
		POptix::LightSample lightSample; // Sample one of many lights.
		lightSample.index = optix::clamp(static_cast<int>(floorf(rng(thePrd.seed) * sysNumberOfLights)), 0, sysNumberOfLights - 1);
		POptix::Light light = sysLightParameters[lightSample.index];
		const POptix::ELightType lightType = light.lightType;

		sysLightSample[lightType](light, thePrd, lightSample, state); // lightSample direction and distance returned in world space!

		rtPrintf("Light Index : %d \n", lightSample.index);
		rtPrintf("Light pdf : %f \n", lightSample.pdf);

		if (lightSample.pdf > 0.0f) // Useful light sample?
		{
			float3 feval = make_float3(0.0f);
			float pdf = 0.0f;

			if (thePrd.brdf_flags & POptix::BSDF_DIFFUSE)
			{
				// Diffuse evaluation
				// Evaluate the Lambert BSDF in the light sample direction. Normally cheaper than shooting rays.
				sysBRDFPdf[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
				feval = sysBRDFEval[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
				pdf = thePrd.pdf;
			}
			else if (thePrd.brdf_flags & POptix::BSDF_GLOSSY)
			{
				// Specular evaluation
				sysBRDFPdf[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
				feval = sysBRDFEval[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
				pdf = thePrd.pdf;
			}

			
			rtPrintf("PDF : %f \n", pdf);

			if (0.0f < pdf && isNotNull(feval))
			{
				// Do the visibility check of the light sample.
				ShadowPRD prdShadow;
				prdShadow.visible = true; // Initialize for miss.

				// Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval 
				// to prevent self intersections with the actual light geometry in the scene!
				rtPrintf("lightSample.direction : %f, %f, %f\n", lightSample.direction.x, lightSample.direction.y, lightSample.direction.z);
				rtPrintf("Light distance : %f \n", lightSample.distance);
				optix::Ray ray = optix::make_Ray(thePrd.hit_pos, lightSample.direction, 1, sysSceneEpsilon, lightSample.distance - sysSceneEpsilon); // Shadow ray.
				rtTrace(sysTopObject, ray, prdShadow);

				if (prdShadow.visible)
				{
					//float NdotL = dot(shading_normal, -lightSample.direction);
					//float lightPdf = lightSample.pdf < 0.0f ? (lightSample.distance * lightSample.distance) / (light.area * NdotL) : lightSample.pdf;

					float lightPdf = lightSample.pdf;
					rtPrintf("Light PDF : %f \n", lightPdf);
					const float misWeight = powerHeuristic(lightPdf, pdf);
					thePrd.radiance += feval * lightSample.emission * (misWeight * fabsf(optix::dot(lightSample.direction, shading_normal)) / lightPdf);
				}
			}
		}
	}
#endif // USE_NEXT_EVENT_ESTIMATION
}

RT_PROGRAM void any_hit()
{
	prd_shadow.visible = false;
	rtTerminateRay();

	thePrd.flags |= FLAG_TERMINATE;
}
