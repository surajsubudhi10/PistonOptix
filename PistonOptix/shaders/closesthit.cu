#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, parMaterialIndex, , ); // Per Material index into the sysMaterialParameters array.
rtDeclareVariable(int, programId, , );

rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFEval;


// Helper functions for sampling a cosine weighted hemisphere distrobution as needed for the Lambert shading model.

RT_FUNCTION void alignVector(float3 const& axis, float3& w)
{
	// Align w with axis.
	const float s = copysign(1.0f, axis.z);
	w.z *= s;
	const float3 h = make_float3(axis.x, axis.y, axis.z + s);
	const float  k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
	w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const& axis, float3& w, float& pdf)
{
	// Choose a point on the local hemisphere coordinates about +z.
	const float theta = 2.0f * M_PIf * sample.x;
	const float r = sqrtf(sample.y);
	w.x = r * cosf(theta);
	w.y = r * sinf(theta);
	w.z = 1.0f - w.x * w.x - w.y * w.y;
	w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

	pdf = w.z * M_1_PIf;

	// Align with axis.
	alignVector(axis, w);
}


RT_PROGRAM void closesthit()
{
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	// Advance the path to the hit position in world coordinates.
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance; 

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		shading_normal = -shading_normal;
		// Do not recalculate the frontface condition!
	}

	State state;
	state.hit_position = thePrd.hit_pos;
	state.shading_normal = shading_normal;

	// A material system with support for arbitrary mesh lights would evaluate its emission here.
	thePrd.radiance = make_float3(0.0f);

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	MaterialParameter mat = sysMaterialParameters[parMaterialIndex];


	// BRDF Sampling
	sysBRDFSample[0](mat, state, thePrd);
	sysBRDFPdf[0](mat, state, thePrd);
	float3 f = sysBRDFEval[0](mat, state, thePrd);

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	// PERF Since the cosine-weighted hemisphere distribution is a perfect importance-sampling of the Lambert material,
	// the whole term ((M_1_PIf * fabsf(optix::dot(prd.wi, normal)) / prd.pdf) is always 1.0f here!
	thePrd.f_over_pdf = f * fabsf(optix::dot(thePrd.wi, state.shading_normal)) / thePrd.pdf;

	// This is a brute-force path tracer. There is no next event estimation (direct lighting) here.
	// Note that because of that, the albedo affects the path throughput only.
	// This material is not returning any radiance because it's not a light source.
}
