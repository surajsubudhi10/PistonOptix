#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"
#include "PistonOptix/inc/LightParameters.h"


// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );
rtDeclareVariable(ShadowPRD, prd_shadow, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent, attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord, attribute TEXCOORD, );

// POptix::Material parameter definition.
rtBuffer<POptix::Material> sysMaterialParameters; // Context global buffer with an array of structures of POptix::Material.
rtDeclareVariable(int, parMaterialIndex, , ); // Per POptix::Material index into the sysMaterialParameters array.
rtDeclareVariable(int, programId, , );

rtBuffer< rtCallableProgramId<void(POptix::Material &mat, State &state, PerRayData &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(POptix::Material &mat, State &state, PerRayData &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(POptix::Material &mat, State &state, PerRayData &prd)> > sysBRDFEval;

rtBuffer< rtCallableProgramId<void(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)> > sysLightSample;
rtBuffer<POptix::Light> sysLightParameters;
rtDeclareVariable(int, sysNumberOfLights, , );

RT_FUNCTION float3 DirectLighting(POptix::Material &mat, State& state);

RT_PROGRAM void closesthit()
{
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	// Advance the path to the hit position in world coordinates.
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance;

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		shading_normal = -shading_normal;
	}

	State state;
	state.hit_position = thePrd.hit_pos;
	state.shading_normal = shading_normal;
	state.geometry_normal = geoNormal;

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.radiance = make_float3(0.0f);
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	POptix::Material mat = sysMaterialParameters[parMaterialIndex];
	float3 baseColor = mat.albedo;
	float metallic = mat.metallic;

	float3 diffuseBRDF = make_float3(0.0f);
	float3 specularBRDF = make_float3(0.0f);

	// Roulette-select the ray's path
	float roulette = rng(thePrd.seed);
	float diffChance = 0.5f * (1.0f - mat.metallic);
	if (roulette < diffChance)
	{
		// Diffuse reflection
		sysBRDFSample[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
		sysBRDFPdf[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
		diffuseBRDF = sysBRDFEval[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);

		thePrd.brdf_flags |= POptix::BSDF_REFLECTION;
		thePrd.brdf_flags |= POptix::BSDF_DIFFUSE;
	}
	else
	{
		// Specular reflection
		sysBRDFSample[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		sysBRDFPdf[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		specularBRDF = sysBRDFEval[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);

		thePrd.brdf_flags |= POptix::BSDF_REFLECTION;
		thePrd.brdf_flags |= (mat.roughness > 0.0f) ? POptix::BSDF_GLOSSY : POptix::BSDF_SPECULAR;
	}

	float3 wiWorld = thePrd.wi;
	float3 woWorld = -theRay.direction;
	float3 H = normalize(woWorld + wiWorld);

	float3 dielectricSpecular = make_float3(0.04f, 0.04f, 0.04f);
	float3 F0 = lerp(dielectricSpecular, baseColor, metallic);
	float3 F = F0 + (1.0f - F0) * powf(1.0f - dot(wiWorld, H), 5.0f);
	float3 f = (1.0f - F) * diffuseBRDF + specularBRDF;

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	thePrd.f_over_pdf = f * fabsf(optix::dot(thePrd.wi, state.shading_normal)) / thePrd.pdf;

#if USE_NEXT_EVENT_ESTIMATION
	thePrd.radiance += DirectLighting(mat, state);
#endif // USE_NEXT_EVENT_ESTIMATION
}

RT_PROGRAM void any_hit()
{
	prd_shadow.visible = false;
	rtTerminateRay();

	thePrd.flags |= FLAG_TERMINATE;
}

RT_FUNCTION float3 DirectLighting(POptix::Material &mat, State& state)
{
	float3 result = make_float3(0.0f);
	if ((thePrd.brdf_flags & (POptix::BSDF_DIFFUSE | POptix::BSDF_GLOSSY)) && sysNumberOfLights > 0)
	{
		// Setp 1: Sample one of many lights.
		POptix::LightSample lightSample;

		int lightNum = min((int)(rng(thePrd.seed) * sysNumberOfLights), sysNumberOfLights - 1);
		float lightPdf = 1.0f / sysNumberOfLights;
		POptix::Light sampledlight = sysLightParameters[lightNum];

		// Step 2: lightSample direction and distance and directLightPDF returned in world space!
		sysLightSample[sampledlight.lightType](sampledlight, thePrd, lightSample, state);

		float3 Ld = make_float3(0.0f);
		float3 Li = make_float3(0.0f);
		// Sample light source with multiple importance sampling
		float directLightPdf = 0.0f;
		float scatteringPdf = 0.0f;

		if (lightSample.pdf == 0 || lightSample.distance == 0) 
		{
			return make_float3(0.0f);
		}

		if (dot(sampledlight.normal, -lightSample.direction) > 0.0f)
		{
			Li = lightSample.emission;
			directLightPdf = lightSample.pdf;
		}

		if (lightSample.pdf > 0.0f && isNotNull(Li))
		{
			// Step 3: Compute BSDF value for light sample
			float3 f = make_float3(0.0f);
			if (thePrd.brdf_flags & POptix::BSDF_DIFFUSE)
			{
				// Diffuse evaluation
				sysBRDFPdf[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
				f = sysBRDFEval[POptix::EBrdfTypes::LAMBERT](mat, state, thePrd);
				scatteringPdf = thePrd.pdf;
			}
			else if (thePrd.brdf_flags & POptix::BSDF_GLOSSY)
			{
				// Specular evaluation
				sysBRDFPdf[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
				f = sysBRDFEval[POptix::EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
				scatteringPdf = thePrd.pdf;
			}

			if (isNotNull(f)) 
			{
				// Do the visibility check of the light sample.
				ShadowPRD prdShadow;
				prdShadow.visible = true; // Initialize for miss.

				// Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval 
				// to prevent self intersections with the actual light geometry in the scene!
				float3 lightDir = normalize(lightSample.direction);
				optix::Ray ray = optix::make_Ray(thePrd.hit_pos, lightDir, 1, sysSceneEpsilon, lightSample.distance - sysSceneEpsilon); // Shadow ray.
				rtTrace(sysTopObject, ray, prdShadow);

				if (prdShadow.visible)
				{
					// Add light's contribution to reflected radiance
					if (sampledlight.isDelta)
					{
						Ld += f * Li / directLightPdf;
					}
					else 
					{
						float weight = PowerHeuristic(1.f, directLightPdf, 1.0f, scatteringPdf);
						Ld += f * Li * weight / directLightPdf;
					}
				}
			}
		}
		result = Ld / lightPdf;
	}

	return result;
}
