#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, parMaterialIndex, , ); // Per Material index into the sysMaterialParameters array.
rtDeclareVariable(int, programId, , );

rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFPdf;
rtBuffer< rtCallableProgramId<void(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFSample;
rtBuffer< rtCallableProgramId<float3(MaterialParameter &mat, State &state, PerRayData &prd)> > sysBRDFEval;


RT_FUNCTION float sdot(float3 x, float3 y)
{
	return clamp(dot(x, y), 0.0f, 1.0f);
}

RT_FUNCTION float SmoothnessToPhongAlpha(float s)
{
	return pow(1000.0f, s * s);
}


RT_PROGRAM void closesthit()
{
	float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
	float3 shading_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

	// Advance the path to the hit position in world coordinates.
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance; 

	// Explicitly include edge-on cases as frontface condition! (Important for nested materials shown in a later example.)
	thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, geoNormal)) ? FLAG_FRONTFACE : 0;

	if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
	{
		// Means geometric normal and shading normal are always defined on the side currently looked at.
		// This gives the backfaces of opaque BSDFs a defined result.
		geoNormal = -geoNormal;
		shading_normal = -shading_normal;
		// Do not recalculate the frontface condition!
	}

	State state;
	state.hit_position = thePrd.hit_pos;
	state.shading_normal = shading_normal;

	// A material system with support for arbitrary mesh lights would evaluate its emission here.
	thePrd.radiance = make_float3(0.0f);

	// Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
	thePrd.f_over_pdf = make_float3(0.0f);
	thePrd.pdf = 0.0f;

	MaterialParameter mat = sysMaterialParameters[parMaterialIndex];

	float3 baseColor = mat.albedo;
	float metallic = mat.metallic;

	float3 dielectricSpecular = make_float3(0.04f, 0.04f, 0.04f);
	float3 F0 = lerp(dielectricSpecular, baseColor, metallic);

	float3 diffuseBRDF = make_float3(0.0f);
	float3 specularBRDF = make_float3(0.0f);

	float diffChance = intensity(baseColor);

	// Roulette-select the ray's path
	float roulette = rng(thePrd.seed);
	if (roulette < diffChance) 
	{
		// Diffuse reflection
		sysBRDFSample[EBrdfTypes::LAMBERT](mat, state, thePrd);
		sysBRDFPdf[EBrdfTypes::LAMBERT](mat, state, thePrd);
		diffuseBRDF = sysBRDFEval[EBrdfTypes::LAMBERT](mat, state, thePrd);
	}
	else
	{
		// Specular reflection
		sysBRDFSample[EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		sysBRDFPdf[EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
		specularBRDF = sysBRDFEval[EBrdfTypes::MICROFACET_REFLECTION](mat, state, thePrd);
	}

	float3 wiWorld = thePrd.wi;
	float3 woWorld = -theRay.direction;
	//float3 N = state.shading_normal;
	float3 H = normalize(woWorld + wiWorld);


	float3 F = F0 + (1.0f - F0) * powf(1.0f - dot(wiWorld, H), 5.0f);
	float3 f = (1.0f - F) * diffuseBRDF + specularBRDF;
	

	// Do not sample opaque surfaces below the geometry!
	// Mind that the geometry normal has been flipped to the side the ray points at.
	if (thePrd.pdf <= 0.0f || optix::dot(thePrd.wi, geoNormal) <= 0.0f)
	{
		thePrd.flags |= FLAG_TERMINATE;
		return;
	}

	
	thePrd.f_over_pdf = f * fabsf(optix::dot(thePrd.wi, state.shading_normal)) / thePrd.pdf;
}
