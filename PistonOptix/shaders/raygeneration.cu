#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "shader_common.h"

#include "rt_assert.h"

rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );
rtDeclareVariable(int2, sysPathLengths, , );
rtDeclareVariable(int, sysIterationIndex, , );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

using namespace optix;

#if !USE_SHADER_TONEMAP
rtDeclareVariable(float, invWhitePoint, , );
rtDeclareVariable(float3, colorBalance, , );
rtDeclareVariable(float, burnHighlights, , );
rtDeclareVariable(float, saturation, , );
rtDeclareVariable(float, crushBlacks, , );
rtDeclareVariable(float, invGamma, , );
rtDeclareVariable(int, useToneMapper, , );


RT_FUNCTION float3 ToneMap(optix::float3& hdrColor)
{
	if(useToneMapper == 0)
	{
		return hdrColor;
	}

	float3 ldrColor = invWhitePoint * colorBalance * hdrColor;
	ldrColor *= (ldrColor * burnHighlights + 1.0) / (ldrColor + 1.0);
	float luminance = dot(ldrColor, make_float3(0.3, 0.59, 0.11));
	ldrColor = fmaxf(lerp(make_float3(luminance), ldrColor, saturation), 0.0f);
	luminance = dot(ldrColor, make_float3(0.3, 0.59, 0.11));
	if (luminance < 1.0)
	{
	  ldrColor = fmaxf(lerp(fpowf(ldrColor, make_float3(crushBlacks)), ldrColor, sqrt(luminance)), 0.0f);
	}

	ldrColor = fpowf(ldrColor, make_float3(invGamma));
	return ldrColor;
}
#endif


RT_FUNCTION void integrator(PerRayData& prd, float3& radiance)
{
	radiance = make_float3(0.0f);				// Start with black.
	float3 throughput = make_float3(1.0f);		// The throughput for the next radiance, starts with 1.0f.
	int depth = 0;								// Path segment index. Primary ray is 0.

	while (depth < sysPathLengths.y)
	{
		prd.wo = -prd.wi;						// wi is the next path segment ray.direction. wo is the direction to the observer.
		prd.flags = 0;							// Clear all non-persistent flags. None in this version.

		// Note that the primary rays wouldn't need to offset the ray t_min by sysSceneEpsilon.
		optix::Ray ray = optix::make_Ray(prd.hit_pos, prd.wi, 0, sysSceneEpsilon, RT_DEFAULT_MAX);
		rtTrace(sysTopObject, ray, prd);

		radiance += throughput * prd.radiance;

		// Path termination by miss shader or sample() routines.
		// If terminate is true, f_over_pdf and pdf might be undefined.
		if ((prd.flags & FLAG_TERMINATE) || prd.pdf <= 0.0f || isNull(prd.f_over_pdf))
		{
			break;
		}

		// PERF f_over_pdf already contains the proper throughput adjustment for diffuse materials: f * (fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);
		throughput *= prd.f_over_pdf;

		// Russian Roulette path termination after a specified number of bounces in sysPathLengths.x would go here. See next examples.

		++depth; // Next path segment.
	}
}

// Entry point for pinhole camera with manual accumulation, non-VCA.
RT_PROGRAM void raygeneration()
{
	PerRayData prd;

	// Initialize the random number generator seed from the linear pixel index and the iteration index.
	prd.seed = tea<8>(theLaunchIndex.y * theLaunchDim.x + theLaunchIndex.x, sysIterationIndex);

	// Pinhole camera implementation:
	// The launch index is the pixel coordinate.
	// Note that launchIndex = (0, 0) is the bottom left corner of the image,
	// which matches the origin in the OpenGL texture used to display the result.
	const float2 pixel = make_float2(theLaunchIndex);
	// Sample the ray in the center of the pixel.
	const float2 fragment = pixel + rng2(prd.seed); // Random jitter of the fragment location in this pixel.
	// The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
	const float2 screen = make_float2(theLaunchDim);
	// Normalized device coordinates in range [-1, 1].
	const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

	// The integrator expects the next path segments ray.origin in prd.pos and the next ray.direction in prd.wi.
	prd.hit_pos = sysCameraPosition;
	prd.wi = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

	float3 radiance;

	integrator(prd, radiance); // In this case a unidirectional path tracer.

#if !USE_SHADER_TONEMAP
	radiance = ToneMap(radiance);
#endif

#if USE_DEBUG_EXCEPTIONS
  // DAR DEBUG Highlight numerical errors.
	if (isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z))
	{
		radiance = make_float3(1000000.0f, 0.0f, 0.0f); // super red
	}
	else if (isinf(radiance.x) || isinf(radiance.y) || isinf(radiance.z))
	{
		radiance = make_float3(0.0f, 1000000.0f, 0.0f); // super green
	}
	else if (radiance.x < 0.0f || radiance.y < 0.0f || radiance.z < 0.0f)
	{
		radiance = make_float3(0.0f, 0.0f, 1000000.0f); // super blue
	}
#else
  // NaN values will never go away. Filter them out before they can arrive in the output buffer.
  // This only has an effect if the debug coloring above is off!
	if (!(isnan(radiance.x) || isnan(radiance.y) || isnan(radiance.z)))
#endif
	{
		if (0 < sysIterationIndex)
		{
			float4 dst = sysOutputBuffer[theLaunchIndex];  // RGBA32F
			sysOutputBuffer[theLaunchIndex] = optix::lerp(dst, make_float4(radiance, 1.0f), 1.0f / (float)(sysIterationIndex + 1));
		}
		else
		{
			// sysIterationIndex 0 will fill the buffer.
			// If this isn't done separately, the result of the lerp() above is undefined, e.g. dst could be NaN.
			sysOutputBuffer[theLaunchIndex] = make_float4(radiance, 1.0f);
		}
	}
}

