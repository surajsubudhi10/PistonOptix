#include "hip/hip_runtime.h"

#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

rtDeclareVariable(Ray, theRay, rtCurrentRay, );

RT_CALLABLE_PROGRAM void PDF(MaterialParameter &mat, State &state, PerRayData &prd)
{
	prd.pdf = 0.5f * M_1_PI; // (1 / 2PI)
}

RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;				// In World Coordinate
	float3 wo = -theRay.direction;					// In World Coordinate (viewer direction)

	float3 dir = UniformHemisphereSampling(rng2(prd.seed));

	OrthoNormBasis shadingONB(N);
	shadingONB.ToBasisCoordinate(wo);				// In Shading Coordinate

	// if the viewer dir is opossite to surface normal (backface)
	if (wo.z < 0.0f)
		dir.z *= -1.0f;

	shadingONB.ToWorldCoordinate(dir);				// In World coordinate
	
	prd.hit_pos = state.hit_position;
	prd.wi = dir;
}


RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData &prd)
{
	return mat.albedo * M_1_PIf;
}
