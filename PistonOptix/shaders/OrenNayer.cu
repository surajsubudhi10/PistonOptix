#include "hip/hip_runtime.h"

#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

rtDeclareVariable(Ray, theRay, rtCurrentRay, );

RT_CALLABLE_PROGRAM void PDF(POptix::Material &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;

	bool sameHemisphere = dot(wiWorld, N) * dot(woWorld, N) > 0 ? true : false;
	prd.pdf = sameHemisphere ? fabsf(dot(wiWorld, N)) * M_1_PIf : 0.0f;			// Importance Sampling
	// prd.pdf = 0.5f * M_1_PI; // (1 / 2PI)									// Uniform Sampling
}

RT_CALLABLE_PROGRAM void Sample(POptix::Material &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)

	float3 dir = UnitSquareToCosineHemisphere(rng2(prd.seed));

	TBN onb(N);
	float3 wo = onb.transform(woWorld);

	if (wo.z < 0.0f)
		dir.z *= -1.0f;

	prd.wi = onb.inverse_transform(dir);
}


RT_CALLABLE_PROGRAM float3 Eval(POptix::Material &mat, State &state, PerRayData &prd)
{
	// https://seblagarde.wordpress.com/2011/08/17/hello-world/
	return mat.albedo * M_1_PIf;
}
