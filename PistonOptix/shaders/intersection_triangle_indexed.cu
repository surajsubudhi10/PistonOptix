#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "rt_function.h"
#include "vertex_attributes.h"

rtBuffer<VertexAttributes> attributesBuffer;
rtBuffer<uint3>            indicesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent, attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord, attribute TEXCOORD, );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, sysSceneEpsilon, , );

using namespace optix;

RT_FUNCTION bool IsInsideTriangle(const float3& v0, const float3& v1, const float3& v2, const float3& P)
{
	float3 edge0 = v1 - v0;
	float3 edge1 = v2 - v1;
	float3 edge2 = v0 - v2;
	float3 N = cross(-edge2, edge0);

	float3 C0 = P - v0;
	float3 C1 = P - v1;
	float3 C2 = P - v2;
	if (dot(N, cross(edge0, C0)) > 0 &&
		dot(N, cross(edge1, C1)) > 0 &&
		dot(N, cross(edge2, C2)) > 0) return true; // P is inside the triangle 

	return false;
}

RT_FUNCTION bool IntersectTriangle(const Ray& ray, const float3& v0, const float3& v1, const float3& v2, float3& N, float& t, float& beta, float& gamma) 
{
	// compute plane's normal
	const float3 ed0 = v1 - v0;
	const float3 ed1 = v0 - v2;
	N = cross(ed1, ed0);

	double a = v0.x - v1.x, b = v0.x - v2.x, c = ray.direction.x, d = v0.x - ray.origin.x;
	double e = v0.y - v1.y, f = v0.y - v2.y, g = ray.direction.y, h = v0.y - ray.origin.y;
	double i = v0.z - v1.z, j = v0.z - v2.z, k = ray.direction.z, l = v0.z - ray.origin.z;

	double m = f * k - g * j, n = h * k - g * l, p = f * l - h * j;
	double q = g * i - e * k, s = e * j - f * i;
	double inv_denom = 1.0 / (a * m + b * q + c * s);
	double e1 = d * m - b * n - c * p;
	
	beta = e1 * inv_denom;
	if (beta < 0.0)
		return (false);
	
	double r = r = e * l - h * i;
	double e2 = a * n + d * q + c * r;
	gamma = e2 * inv_denom;
	if (gamma < 0.0)
		return (false);
	if (beta + gamma > 1.0)
		return false;
	
	double e3 = a * p - b * r + d * s;
	t = e3 * inv_denom;
	if (t < ray.tmin && t > ray.tmax)
		return false;
	
	return true;
}

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex)
{
	const uint3 indices = indicesBuffer[primitiveIndex];

	VertexAttributes const& a0 = attributesBuffer[indices.x];
	VertexAttributes const& a1 = attributesBuffer[indices.y];
	VertexAttributes const& a2 = attributesBuffer[indices.z];

	const float3 v0 = a0.vertex;
	const float3 v1 = a1.vertex;
	const float3 v2 = a2.vertex;

	float3 n;
	float  t;
	float  beta;
	float  gamma;

	//if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
	if (IntersectTriangle(theRay, v0, v1, v2, n, t, beta, gamma))
	{
		if (rtPotentialIntersection(t))
		{
			// Barycentric interpolation:
			const float alpha = 1.0f - beta - gamma;

			// Note: No normalization on the TBN attributes here for performance reasons.
			//       It's done after the transformation into world space anyway.
			varGeoNormal = n;
			varTangent = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
			varNormal = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
			varTexCoord = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;

			rtReportIntersection(0);
		}
	}
}
