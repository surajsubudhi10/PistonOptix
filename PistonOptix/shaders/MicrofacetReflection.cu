#include "hip/hip_runtime.h"

#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

rtDeclareVariable(Ray, theRay, rtCurrentRay, );


RT_FUNCTION float smithG_GGX(float NDotv, float alphaG)
{
	float a = alphaG * alphaG;
	float b = NDotv * NDotv;
	return 1.0f / (NDotv + sqrtf(a + b - a * b));
}

RT_FUNCTION float TrowbridgeReitzDistribution_D(float cosTheta, float alpha)
{
	float Cos2Theta = cosTheta * cosTheta;
	float Sin2Theta = 1.0f - Cos2Theta;
	float tan2Theta = Sin2Theta / Cos2Theta;

	if (tan2Theta > 10e12) 
		return 0.0f;

	const float cos4Theta = Cos2Theta * Cos2Theta;
	float e = (1.0f / (alpha * alpha)) * tan2Theta;
	return 1.0f * charFunc(cosTheta) / (M_PIf * alpha * alpha * cos4Theta * (1 + e) * (1 + e));
}

RT_FUNCTION float TrowbridgeReitzDistribution_Lambda(float cosTheta, float alpha)
{
	float Cos2Theta = cosTheta * cosTheta;
	float SinTheta = sqrtf(1.0f - Cos2Theta);

	float absTanTheta = abs(SinTheta / cosTheta);
	if (isinf(absTanTheta))
		return 0.;

	float alpha2Tan2Theta = (alpha * absTanTheta) * (alpha * absTanTheta);
	return (-1 + sqrt(1.f + alpha2Tan2Theta)) / 2;
}

RT_FUNCTION float TrowbridgeReitzDistribution_G(const float3& vec, const float3& halfVec, const float3& normal, float alpha)
{
	float vDotH = dot(vec, halfVec);
	float vDotN = dot(vec, normal);

	float tan2V = (1.0f - (vDotN * vDotN)) / (vDotN * vDotN);
	float mult = charFunc(vDotH / vDotN);
	float deno = 1.0f + sqrt(1.0f + alpha * alpha * tan2V);

	return mult * 2.0f / deno;
}

RT_FUNCTION float TrowbridgeReitzDistribution_RoughnessToAlpha(float roughness) 
{
	roughness = max(roughness, (float)1e-3);
	float x = log(roughness);
	return 1.62142f + 0.819955f * x + 0.1734f * x * x + 0.0171201f * x * x * x + 0.000640711f * x * x * x * x;
}



RT_CALLABLE_PROGRAM void PDF(MaterialParameter &mat, State &state, PerRayData &prd)
{
	/*
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;
	float3 H = normalize(wiWorld + woWorld);

	float WoDotH = dot(woWorld, H);
	float cosThetaH = dot(H, N);
	float cosThetaO = (dot(woWorld, N));

	float alpha = TrowbridgeReitzDistribution_RoughnessToAlpha(mat.roughness);
	float D = TrowbridgeReitzDistribution_D(cosThetaH, alpha);
	//float G1 = 1.0f / (1.0f + TrowbridgeReitzDistribution_Lambda(cosThetaO, alpha));

	float pdf = D * G1 * abs(WoDotH) / 4.0f * abs(cosThetaO) * WoDotH;

	bool sameHemisphere = dot(wiWorld, H) * dot(woWorld, H) > 0 ? true : false;
	prd.pdf = sameHemisphere ? pdf : 0.0f;			// Importance Sampling
	*/

	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;
	float3 H = normalize(wiWorld + woWorld);

	float cosTheta = dot(wiWorld, H);
	float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
	
	float alpha = powf(max(0.001f, mat.roughness), 2.0f);
	float D = TrowbridgeReitzDistribution_D(cosTheta, alpha);

	float pdf = D * cosTheta * sinTheta;

	bool sameHemisphere = dot(wiWorld, H) * dot(woWorld, H) > 0 ? true : false;
	prd.pdf = sameHemisphere ? pdf : 0.0f;			// Importance Sampling

}

RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	
	float2 r = rng2(prd.seed);

	optix::Onb onb(N); // basis
	float alpha = powf(max(0.001f, mat.roughness), 2.0f);
	
	float phi = r.x * 2.0f * M_PIf;
	float cosTheta = sqrtf((1.0f - r.y) / (1.0f + (alpha*alpha - 1.0f) * r.y));
	float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
	float sinPhi = sinf(phi);
	float cosPhi = cosf(phi);

	float3 half = make_float3(sinTheta*cosPhi, sinTheta*sinPhi, cosTheta);
	onb.inverse_transform(half);
	//AlignVector(N, half);
	float3 dir = 2.0f*dot(woWorld, half)*half - woWorld; //reflection vector

	prd.wi = dir;
}

RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;
	float3 H = normalize(wiWorld + woWorld);

	float cosThetaO = fabsf(dot(woWorld, N));
	float cosThetaI = fabsf(dot(wiWorld, N));

	if (cosThetaI <= 0.0f || cosThetaO <= 0.0f) 
	{
		prd.flags |= FLAG_TERMINATE;
		return make_float3(0.0f);
	}

	if (H.x == 0 && H.y == 0 && H.z == 0) 
	{
		prd.flags |= FLAG_TERMINATE;
		return make_float3(0.0f);
	}

	float3 dielectricSpecular = make_float3(0.04f, 0.04f, 0.04f);
	float3 F0 = lerp(dielectricSpecular, mat.albedo, mat.metallic);
	float3 F = F0 + (1.0f - F0) * powf(1.0f - dot(wiWorld, H), 5.0f);

	float cosThetaH = dot(H, N);

	float alpha = powf(max(0.001f, mat.roughness), 2.0f);
	float D = TrowbridgeReitzDistribution_D(cosThetaI, alpha);
	float G = TrowbridgeReitzDistribution_G(woWorld, H, N, alpha) * TrowbridgeReitzDistribution_G(wiWorld, H, N, alpha);

	return F * G * D;// / (4.0f * fabsf(cosThetaI) * fabsf(cosThetaO));
}
