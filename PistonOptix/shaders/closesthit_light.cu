#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"
#include "PistonOptix/inc/LightParameters.h"


// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );

rtBuffer<POptix::Light> sysLightParameters;
rtDeclareVariable(int, parMaterialIndex, , );  // Index into the sysLightDefinitions array.
rtDeclareVariable(int, sysNumberOfLights, , );


RT_PROGRAM void closesthit_light()
{
	thePrd.hit_pos = theRay.origin + theRay.direction * theIntersectionDistance;

	// PERF Not really needed when it's know that light geometry is not under Transforms.
	const float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));

	const float cosTheta = optix::dot(thePrd.wo, geoNormal);
	thePrd.flags |= (0.0f <= cosTheta) ? FLAG_FRONTFACE : 0;

	thePrd.radiance = make_float3(0.0f); // Backside is black.

	if (thePrd.flags & FLAG_FRONTFACE) // Looking at the front face?
	{
		const POptix::Light light = sysLightParameters[parMaterialIndex];
		thePrd.radiance = light.emission;

#if USE_NEXT_EVENT_ESTIMATION
		const float pdfLight = (theIntersectionDistance * theIntersectionDistance) / (light.area * cosTheta);
		// If it's an implicit light hit from a diffuse scattering event and the light emission was not returning a zero pdf.
		if ((thePrd.brdf_flags & (POptix::BSDF_DIFFUSE | POptix::BSDF_GLOSSY)) && DENOMINATOR_EPSILON < pdfLight)
		{
			// Scale the emission with the power heuristic between the previous BSDF sample pdf and this implicit light sample pdf.
			thePrd.radiance *= powerHeuristic(thePrd.pdf, pdfLight);
		}
#endif // USE_NEXT_EVENT_ESTIMATION
	}

	// Lights have no other material properties than emission in this demo. Terminate the path.
	thePrd.flags |= FLAG_TERMINATE;
}
