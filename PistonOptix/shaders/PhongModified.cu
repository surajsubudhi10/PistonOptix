#include "hip/hip_runtime.h"

#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

rtDeclareVariable(Ray, theRay, rtCurrentRay, );

RT_CALLABLE_PROGRAM void PDF(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;

	float cosTheta = dot(wiWorld, N);
	float alpha = mat.roughness;

	bool sameHemisphere = cosTheta * dot(woWorld, N) > 0 ? true : false;
	prd.pdf = sameHemisphere ? satu(powf(fabsf(cosTheta), alpha)) * M_2_PIf * (alpha + 1.0f) : 0.0f;			// Importance Sampling
}

RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)

	float3 wiWorld = reflect(-woWorld, N);
	float3 dir = CosineWeightedHemisphereSampling(rng2(prd.seed), mat.roughness);

	AlignVector(N, dir);

	TBN onb(N);
	prd.wi = /*wiWorld;*/ dir;
	//prd.wi = onb.inverse_transform(dir);
}


RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;

	float cosTheta = dot(wiWorld, N);
	float alpha = mat.roughness;

	// https://seblagarde.wordpress.com/2011/08/17/hello-world/
	return mat.albedo * M_2_PIf * (alpha + 2.0f) * satu(powf(cosTheta, alpha));
}
