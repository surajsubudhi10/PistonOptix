#include "hip/hip_runtime.h"

#include "per_ray_data.h"
#include "material_parameter.h"
#include "shader_common.h"
#include "PistonOptix/inc/CudaUtils/State.h"

rtDeclareVariable(Ray, theRay, rtCurrentRay, );


RT_FUNCTION float smithG_GGX(float NDotv, float alphaG)
{
	float a = alphaG * alphaG;
	float b = NDotv * NDotv;
	return 1.0f / (NDotv + sqrtf(a + b - a * b));
}

RT_CALLABLE_PROGRAM void PDF(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;
	float3 H = normalize(wiWorld + woWorld);

	float cosTheta = dot(wiWorld, H);
	float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
	float alpha = powf(max(0.001f, mat.roughness), 2.0f);
	float alphaSqr = alpha * alpha;
	
	float pdf = alphaSqr * cosTheta * sinTheta / (M_PIf * powf(cosTheta * cosTheta * (alphaSqr - 1.0f) + 1.0f, 2.0f));

	bool sameHemisphere = cosTheta * dot(woWorld, H) > 0 ? true : false;
	prd.pdf = sameHemisphere ? pdf : 0.0f;			// Importance Sampling
}

RT_CALLABLE_PROGRAM void Sample(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	
	float2 r = rng2(prd.seed);

	optix::Onb onb(N); // basis
	float alpha = powf(max(0.001f, mat.roughness), 2.0f);
	float phi = r.x * 2.0f * M_PIf;

	float cosTheta = sqrtf((1.0f - r.y) / (1.0f + (alpha*alpha - 1.0f) * r.y));
	float sinTheta = sqrtf(1.0f - (cosTheta * cosTheta));
	float sinPhi = sinf(phi);
	float cosPhi = cosf(phi);

	float3 half = make_float3(sinTheta*cosPhi, sinTheta*sinPhi, cosTheta);
	onb.inverse_transform(half);
	//AlignVector(N, half);
	float3 dir = 2.0f*dot(woWorld, half)*half - woWorld; //reflection vector

	prd.wi = dir;
}

RT_CALLABLE_PROGRAM float3 Eval(MaterialParameter &mat, State &state, PerRayData &prd)
{
	float3 N = state.shading_normal;					// In World Coordinate
	float3 woWorld = -theRay.direction;					// In World Coordinate (viewer direction)
	float3 wiWorld = prd.wi;
	float3 H = normalize(wiWorld + woWorld);

	float cosTheta = dot(wiWorld, N);
	float alpha = powf(max(0.001f, mat.roughness), 2.0f);
	float alphaSqr = alpha * alpha;

	float3 dielectricSpecular = make_float3(0.04f, 0.04f, 0.04f);
	float3 F0 = lerp(dielectricSpecular, mat.albedo, mat.metallic);
	float3 F = F0 + (1.0f - F0) * powf(1.0f - dot(wiWorld, H), 5.0f);

	float NDotL = dot(N, wiWorld);
	float NDotV = dot(N, woWorld);

	float vis = 0.5f / (NDotL * sqrt(NDotV * NDotV * (1.0f - alphaSqr) + alphaSqr) + NDotV * sqrt(NDotL * NDotL * (1.0f - alphaSqr) + alphaSqr));
	
	float D =  alphaSqr / (M_PIf * powf(cosTheta * cosTheta * (alphaSqr - 1.0f) + 1.0f, 2.0f));

	// https://blog.selfshadow.com/publications/s2012-shading-course/burley/s2012_pbs_disney_brdf_notes_v3.pdf (Section 5.6)
	float roughg = powf(mat.roughness*0.5f + 0.5f, 2.0f);
	float G = smithG_GGX(NDotL, roughg) * smithG_GGX(NDotV, roughg);

	return F * G * D;
}
