#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "..\inc\CudaUtils\State.h"
#include "..\inc\LightParameters.h"

#include "rt_assert.h"

rtDeclareVariable(int, sysNumberOfLights, , );

RT_FUNCTION float3 UniformSampleSphere(float u1, float u2)
{
	float z = 1.f - 2.f * u1;
	float r = sqrtf(max(0.f, 1.f - z * z));
	float phi = 2.f * M_PIf * u2;
	float x = r * cosf(phi);
	float y = r * sinf(phi);

	return make_float3(x, y, z);
}

RT_CALLABLE_PROGRAM void sphere_sample(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)
{
	const float r1 = rng(prd.seed);
	const float r2 = rng(prd.seed);
	sample.surfacePos = light.position;// +UniformSampleSphere(r1, r2) * light.radius;
	sample.direction = normalize(sample.surfacePos - state.hit_position);
	rtPrintf("sample.direction : %f, %f, %f\n", sample.direction.x, sample.direction.y, sample.direction.z);
	sample.emission = light.emission * sysNumberOfLights;
	sample.distance = length(light.position - state.hit_position);

	//float NdotL = dot(lightSample.direction, -lightDir);
	float lightDistSq = sample.distance * sample.distance;
	sample.pdf = lightDistSq / (light.area);
}

RT_CALLABLE_PROGRAM void quad_sample(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)
{
	const float r1 = rng(prd.seed);
	const float r2 = rng(prd.seed);
	sample.surfacePos = light.position + light.u * r1 + light.v * r2;
	sample.direction = light.direction;
	sample.emission = light.emission * sysNumberOfLights;
	sample.distance = length(sample.surfacePos - state.hit_position);

	if (DENOMINATOR_EPSILON < sample.distance)
	{
		sample.direction /= sample.distance; // Normalized direction to light.

		const float cosTheta = optix::dot(-sample.direction, light.direction);
		if (DENOMINATOR_EPSILON < cosTheta) // Only emit light on the front side.
		{
			// Explicit light sample, must scale the emission by inverse probabilty to hit this light.
			sample.pdf = (sample.distance * sample.distance) / (light.area * cosTheta); // Solid angle pdf. Assumes light.area != 0.0f.
		}
	}
}

RT_CALLABLE_PROGRAM void directional_sample(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)
{
	sample.direction = light.direction;
	sample.distance = RT_DEFAULT_MAX;
	sample.emission = light.emission * sysNumberOfLights;
	sample.pdf = 1.0f;
}
