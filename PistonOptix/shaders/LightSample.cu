#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "..\inc\LightParameters.h"

#include "rt_assert.h"

rtDeclareVariable(int, sysNumberOfLights, , );

RT_FUNCTION float3 UniformSampleSphere(float u1, float u2)
{
	float z = 1.f - 2.f * u1;
	float r = sqrtf(max(0.f, 1.f - z * z));
	float phi = 2.f * M_PIf * u2;
	float x = r * cosf(phi);
	float y = r * sinf(phi);

	return make_float3(x, y, z);
}

RT_CALLABLE_PROGRAM void sphere_sample(LightParameter &light, PerRayData &prd, LightSample &sample)
{
	const float r1 = rng(prd.seed);
	const float r2 = rng(prd.seed);
	sample.surfacePos = light.position + UniformSampleSphere(r1, r2) * light.radius;
	sample.direction = normalize(sample.surfacePos - light.position);
	sample.emission = light.emission * sysNumberOfLights;
	sample.pdf = -1.0f;
}

RT_CALLABLE_PROGRAM void quad_sample(LightParameter &light, PerRayData &prd, LightSample &sample)
{
	const float r1 = rng(prd.seed);
	const float r2 = rng(prd.seed);
	sample.surfacePos = light.position + light.u * r1 + light.v * r2;
	sample.direction = light.direction;
	sample.emission = light.emission * sysNumberOfLights;
	sample.pdf = -1.0f;
}

RT_CALLABLE_PROGRAM void directional_sample(LightParameter &light, PerRayData &prd, LightSample &sample)
{
	sample.direction = light.direction;
	sample.emission = light.emission * sysNumberOfLights;
	sample.pdf = 1.0f;
}
