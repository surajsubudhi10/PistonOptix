#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "shader_common.h"
#include "..\inc\CudaUtils\State.h"
#include "..\inc\LightParameters.h"

#include "rt_assert.h"

rtDeclareVariable(int, sysNumberOfLights, , );

RT_FUNCTION float3 UniformSampleSphere(float u1, float u2)
{
	float z = 1.f - 2.f * u1;
	float r = sqrtf(max(0.f, 1.f - z * z));
	float phi = 2.f * M_PIf * u2;
	float x = r * cosf(phi);
	float y = r * sinf(phi);

	return make_float3(x, y, z);
}

RT_CALLABLE_PROGRAM void sphere_sample(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)
{
	const float r1 = rng(prd.seed);
	const float r2 = rng(prd.seed);
	sample.surfacePos = light.position;// +UniformSampleSphere(r1, r2) * light.radius;
	sample.direction = normalize(sample.surfacePos - state.hit_position);
	rtPrintf("sample.direction : %f, %f, %f\n", sample.direction.x, sample.direction.y, sample.direction.z);
	sample.emission = light.emission * sysNumberOfLights;
	sample.distance = length(light.position - state.hit_position);

	//float NdotL = dot(lightSample.direction, -lightDir);
	float lightDistSq = sample.distance * sample.distance;
	sample.pdf = lightDistSq / (light.area);
}

RT_CALLABLE_PROGRAM void directional_sample(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state)
{
	sample.direction = -light.normal;
	sample.distance = RT_DEFAULT_MAX;
	sample.emission = light.emission * sysNumberOfLights;
	sample.pdf = 1.0f;
}


RT_CALLABLE_PROGRAM void quad_sample(POptix::Light &light, PerRayData &prd, POptix::LightSample &sample, State& state) 
{
	const float r1 = rng(prd.seed);
	const float r2 = rng(prd.seed);
	
	// position on the area light
	sample.surfacePos = light.position + light.u * r1 + light.v * r2;
	sample.pdf = 1.0f / light.area;

	// light ray direction
	float3 wi = normalize(sample.surfacePos - state.hit_position);
	float distance = length(sample.surfacePos - state.hit_position);

	float cosTheta = fabsf(dot(light.normal, -wi));
	if (cosTheta < DENOMINATOR_EPSILON) 
	{
		sample.pdf = 0.0f;
		return;
	}

	sample.pdf *= (distance * distance) / cosTheta;
	if (sample.pdf > 10E19) 
		sample.pdf = 0.0f;

	sample.distance = distance;
	sample.direction = wi;
	sample.emission = light.emission;
}