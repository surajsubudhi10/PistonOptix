#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"

rtDeclareVariable(PerRayData, thePrd, rtPayload, );


RT_PROGRAM void miss_environment_constant()
{
	thePrd.radiance = make_float3(1.0f); // Constant white emission. No next event estimation (direct lighting).
	thePrd.flags |= FLAG_TERMINATE;    // End of path.
}
