#include "hip/hip_runtime.h"
#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"

rtDeclareVariable(PerRayData, thePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtTextureSampler<float4, 2> envmap;


RT_PROGRAM void miss_environment_constant()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	float3 result = make_float3(tex2D(envmap, u, v));

	thePrd.radiance = result;// make_float3(1.0f); // Constant white emission. No next event estimation (direct lighting).
	thePrd.flags |= FLAG_TERMINATE;    // End of path.
}
